#include <iostream>
#include "nvToolsExt.h"
#include "thrust/device_vector.h"

#include "ac.hxx"
#include "helpers.hxx"

#include <hip/hip_runtime.h>               /* for Gpuinfo */
#include <hip/hip_runtime_api.h>   /* for Gpuinfo */
#include <iomanip>
#include <cstdlib>
#include <fstream>
#include <sstream>
#include "json.hpp"
using json = nlohmann::json;

typedef struct Graph {
  Int   num_nodes;
  Int   node_feat_dim;
  Real* node_feats;

  Int   num_edges;
  Int   edge_feat_dim;
  Real* edge_feats;

  Int* srcs;
  Int* dsts;
} Graph;

void loadGraph(std::string inpath, Graph* d_graph) {

  FILE *ptr;
  ptr = fopen(inpath.c_str(), "rb");
    
  Int num_nodes;
  Int node_feat_dim;
  Int num_edges;
  Int edge_feat_dim;
  
  fread(&num_nodes,     sizeof(Int), 1, ptr);
  fread(&node_feat_dim, sizeof(Int), 1, ptr);
  fread(&num_edges,     sizeof(Int), 1, ptr);
  fread(&edge_feat_dim, sizeof(Int), 1, ptr);
  
  Real* node_feats = (Real*)malloc(num_nodes * node_feat_dim * sizeof(Real));
  Real* edge_feats = (Real*)malloc(num_edges * edge_feat_dim * sizeof(Real));
  Int*  srcs       = (Int*)malloc(num_edges * sizeof(Int));
  Int*  dsts       = (Int*)malloc(num_edges * sizeof(Int));
  
  fread(node_feats, sizeof(Real), num_nodes * node_feat_dim, ptr); 
  fread(edge_feats, sizeof(Real), num_edges * edge_feat_dim, ptr); 
  fread(srcs,       sizeof(Int),  num_edges,                 ptr); 
  fread(dsts,       sizeof(Int),  num_edges,                 ptr); 

  // -------------------------
  // Build graph

  d_graph->num_nodes     = num_nodes;
  d_graph->num_edges     = num_edges;
  d_graph->node_feat_dim = node_feat_dim;
  d_graph->edge_feat_dim = edge_feat_dim;

  hipMalloc((void**)&d_graph->node_feats, num_nodes * node_feat_dim * sizeof(Real));
  hipMalloc((void**)&d_graph->edge_feats, num_edges * edge_feat_dim * sizeof(Real));
  hipMalloc((void**)&d_graph->srcs,       num_edges                 * sizeof(Int));
  hipMalloc((void**)&d_graph->dsts,       num_edges                 * sizeof(Int));

  hipMemcpy(d_graph->node_feats, node_feats, num_nodes * node_feat_dim * sizeof(Real), hipMemcpyHostToDevice);
  hipMemcpy(d_graph->edge_feats, edge_feats, num_edges * edge_feat_dim * sizeof(Real), hipMemcpyHostToDevice);
  hipMemcpy(d_graph->srcs,       srcs,       num_edges                 * sizeof(Int), hipMemcpyHostToDevice);
  hipMemcpy(d_graph->dsts,       dsts,       num_edges                 * sizeof(Int), hipMemcpyHostToDevice);
}

json gpu_info_json() {
    json j;
    hipDeviceProp_t devProps;

    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0)   /* no valid devices */
    {
        return j;        /* empty */
    }
    int dev = 0;
    hipGetDevice(&dev);
    hipGetDeviceProperties(&devProps, dev);
    j["gpuinfo"]["name"] = devProps.name;
    j["gpuinfo"]["total_global_mem"] = int64_t(devProps.totalGlobalMem);
    j["gpuinfo"]["major"] = devProps.major;
    j["gpuinfo"]["minor"] = devProps.minor;
    j["gpuinfo"]["clock_rate"] = devProps.clockRate;
    j["gpuinfo"]["multi_processor_count"] = devProps.multiProcessorCount;

    int runtimeVersion, driverVersion;
    hipRuntimeGetVersion(&runtimeVersion);
    hipDriverGetVersion(&driverVersion);
    j["gpuinfo"]["driver_api"] = CUDA_VERSION;
    j["gpuinfo"]["driver_version"] = driverVersion;
    j["gpuinfo"]["runtime_version"] = runtimeVersion;
    j["gpuinfo"]["compute_version"] = devProps.major * 10 + devProps.minor;

    return j;
}

int main ( int argc, char * argv[] ) {

  // --
  // IO

  Graph data;
  Graph patt;
  
  loadGraph(argv[1], &data);
  loadGraph(argv[2], &patt);

  // --
  // Setup GPUs
  
  hipStream_t master_stream;
  hipEvent_t master_event;
  hipStreamCreateWithFlags(&master_stream, hipStreamNonBlocking);
  hipEventCreate(&master_event);

  int _n_gpus = 1;
  hipGetDeviceCount(&_n_gpus);
  Int n_gpus = (Int)(_n_gpus);
  
  for(Int i = 0; i < n_gpus; i++) {
      hipSetDevice(i);
      for(Int j = 0; j < n_gpus; j++) {
          if(i == j) continue;
          hipDeviceEnablePeerAccess(j, 0);
      }
  }
  hipSetDevice(0);
  
  std::vector<gpu_info> infos;
  
  for(Int i = 0 ; i < n_gpus ; i++) {
      gpu_info info;
      hipSetDevice(i);
      hipStreamCreateWithFlags(&info.stream, hipStreamNonBlocking);
      hipEventCreate(&info.event);
      infos.push_back(info);
  }
  hipSetDevice(0);

  Int* chunks    = (Int*)malloc(n_gpus * sizeof(Int));
  Int* starts    = (Int*)malloc(n_gpus * sizeof(Int));
  Int* ends      = (Int*)malloc(n_gpus * sizeof(Int));
  
  for(Int i = 0; i < n_gpus; i++)         chunks[i] = 0;
  for(Int i = 0; i < patt.num_edges; i++) chunks[i % n_gpus]++;
  
  starts[0] = 0;
  ends[0]   = chunks[0];
  for(Int i = 1; i < n_gpus; i++) {
      starts[i] = chunks[i] + starts[i - 1];
      ends[i]   = chunks[i] + ends[i - 1];
  }
  ends[n_gpus - 1] = patt.num_edges;
  
  // --
  // Allocate memory

  Real *MU, *VRmax, *VFmax;
  Real *CV_t, *CE_t, *MU_t, *RE_t, *FE_t, *VR_t, *VF_t, *RMax_t, *FMax_t;

  // Real *Cnull; // Ignoring for now

  hipMalloc((void **)&MU,      data.num_nodes * patt.num_nodes * sizeof(Real));
  hipMalloc((void **)&CV_t,    data.num_nodes * patt.num_nodes * sizeof(Real));
  hipMalloc((void **)&MU_t,    data.num_nodes * patt.num_nodes * sizeof(Real));

  hipMalloc((void **)&VRmax,                    patt.num_edges * sizeof(Real));
  hipMalloc((void **)&VFmax,                    patt.num_edges * sizeof(Real));

  hipMalloc((void **)&CE_t,    data.num_edges * patt.num_edges * sizeof(Real));
  hipMalloc((void **)&RE_t,    data.num_edges * patt.num_edges * sizeof(Real));
  hipMalloc((void **)&FE_t,    data.num_edges * patt.num_edges * sizeof(Real));

  hipMalloc((void **)&VR_t,    data.num_nodes * patt.num_edges * sizeof(Real));
  hipMalloc((void **)&VF_t,    data.num_nodes * patt.num_edges * sizeof(Real));
  hipMalloc((void **)&RMax_t,  data.num_nodes * patt.num_edges * sizeof(Real));
  hipMalloc((void **)&FMax_t,  data.num_nodes * patt.num_edges * sizeof(Real));

  Real* MU_tmp;
  Real* RE_tmp;
  Real* FE_tmp;
  hipMalloc(&MU_tmp, patt.num_nodes * sizeof(Real));
  hipMalloc(&RE_tmp, patt.num_edges * sizeof(Real));
  hipMalloc(&FE_tmp, patt.num_edges * sizeof(Real));

  Real** all_CE_t = (Real**)malloc(n_gpus * sizeof(Real*));
  Real** all_FE_t = (Real**)malloc(n_gpus * sizeof(Real*));
  Real** all_RE_t = (Real**)malloc(n_gpus * sizeof(Real*));
  Real** all_VF_t = (Real**)malloc(n_gpus * sizeof(Real*));
  Real** all_VR_t = (Real**)malloc(n_gpus * sizeof(Real*));
  Real** all_FMax_t = (Real**)malloc(n_gpus * sizeof(Real*));
  Real** all_RMax_t = (Real**)malloc(n_gpus * sizeof(Real*));
  Real** all_VFmax = (Real**)malloc(n_gpus * sizeof(Real*));
  Real** all_VRmax = (Real**)malloc(n_gpus * sizeof(Real*));
  Real** all_FE_tmp = (Real**)malloc(n_gpus * sizeof(Real*));
  Real** all_RE_tmp = (Real**)malloc(n_gpus * sizeof(Real*));
  Int** all_data_srcs = (Int**)malloc(n_gpus * sizeof(Int**));
  Int** all_data_dsts = (Int**)malloc(n_gpus * sizeof(Int**));
  Int** all_patt_srcs = (Int**)malloc(n_gpus * sizeof(Int**));
  Int** all_patt_dsts = (Int**)malloc(n_gpus * sizeof(Int**));

  shard_alloc_n(all_CE_t, n_gpus, patt.num_edges, data.num_edges, starts, ends);

  shard_alloc_n(all_FMax_t, n_gpus, patt.num_edges, data.num_nodes, starts, ends);
  shard_alloc_n(all_RMax_t, n_gpus, patt.num_edges, data.num_nodes, starts, ends);
  
  copy_alloc_n(all_data_srcs, n_gpus, data.num_edges, 1);
  copy_alloc_n(all_data_dsts, n_gpus, data.num_edges, 1);

  copy_alloc_n(all_patt_srcs, n_gpus, patt.num_edges, 1);
  copy_alloc_n(all_patt_dsts, n_gpus, patt.num_edges, 1);
  
  shard_alloc_n(all_FE_t, n_gpus, patt.num_edges, data.num_edges, starts, ends);
  shard_alloc_n(all_RE_t, n_gpus, patt.num_edges, data.num_edges, starts, ends);
  
  shard_alloc_n(all_VF_t, n_gpus, patt.num_edges, data.num_nodes, starts, ends);
  shard_alloc_n(all_VR_t, n_gpus, patt.num_edges, data.num_nodes, starts, ends);

  shard_alloc_n(all_VFmax, n_gpus, patt.num_edges, 1, starts, ends);
  shard_alloc_n(all_VRmax, n_gpus, patt.num_edges, 1, starts, ends);
  
  shard_alloc_n(all_FE_tmp, n_gpus, patt.num_edges, 1, starts, ends);
  shard_alloc_n(all_RE_tmp, n_gpus, patt.num_edges, 1, starts, ends);

  std::vector<row_reducer_t<decltype(hipcub::Max())>> F_max_reducers;
  std::vector<row_reducer_t<decltype(hipcub::Max())>> R_max_reducers;
  std::vector<row_reducer_t<decltype(hipcub::Sum())>> F_sum_reducers;
  std::vector<row_reducer_t<decltype(hipcub::Sum())>> R_sum_reducers;
  // std::vector<row_reducer_t> sum_reducers;
  for(Int gid = 0; gid < n_gpus; gid++) {
    hipSetDevice(gid);
    
    row_reducer_t<decltype(hipcub::Max())> F_max_reducer(
      all_VFmax[gid], all_VF_t[gid], ends[gid] - starts[gid], data.num_nodes, hipcub::Max(), -99999, infos[gid].stream
    );
    row_reducer_t<decltype(hipcub::Max())> R_max_reducer(
      all_VRmax[gid], all_VR_t[gid], ends[gid] - starts[gid], data.num_nodes, hipcub::Max(), -99999, infos[gid].stream
    );
    row_reducer_t<decltype(hipcub::Sum())> F_sum_reducer(
      all_RE_tmp[gid], all_RE_t[gid], ends[gid] - starts[gid], data.num_edges, hipcub::Sum(), 0, infos[gid].stream
    );
    row_reducer_t<decltype(hipcub::Sum())> R_sum_reducer(
      all_FE_tmp[gid], all_FE_t[gid], ends[gid] - starts[gid], data.num_edges, hipcub::Sum(), 0, infos[gid].stream
    );
    
    R_max_reducers.push_back(R_max_reducer);
    F_max_reducers.push_back(F_max_reducer);
    R_sum_reducers.push_back(R_sum_reducer);
    F_sum_reducers.push_back(F_sum_reducer);
  }
  hipSetDevice(0);

  // --
  // Initialize algorithm

  cuda_timer_t timer;
  timer.start();

  cuda_timer_t prep_timer;
  prep_timer.start();

  nvtxRangePushA("start");
  
  nvtxRangePushA("prep");

  ac::cdist(data.num_nodes, patt.num_nodes, patt.node_feat_dim, data.node_feats, patt.node_feats, CV_t);
  ac::cdist(data.num_edges, patt.num_edges, patt.edge_feat_dim, data.edge_feats, patt.edge_feats, CE_t);
  
  thrust::transform(thrust::device, CV_t, CV_t + (patt.num_nodes * data.num_nodes), MU_t, [=] __device__ (Real const& val) {return - val;});
  thrust::transform(thrust::device, CE_t, CE_t + (patt.num_nodes * data.num_nodes), RE_t, [=] __device__ (Real const& val) {return - val;});
  thrust::transform(thrust::device, CE_t, CE_t + (patt.num_nodes * data.num_nodes), FE_t, [=] __device__ (Real const& val) {return - val;});

  ac::RowSoftmax2(patt.num_nodes, data.num_nodes, CV_t, master_stream);
  ac::RowSoftmax2(patt.num_nodes, data.num_nodes, MU_t, master_stream);
  ac::RowSoftmax2(patt.num_edges, data.num_edges, CE_t, master_stream);
  ac::RowSoftmax2(patt.num_edges, data.num_edges, RE_t, master_stream);
  ac::RowSoftmax2(patt.num_edges, data.num_edges, FE_t, master_stream);
  hipDeviceSynchronize();
  
  auto init_VX = [=] __device__(Int const& offset) {
      Int i        = offset % data.num_nodes;
      Int j        = offset / data.num_nodes;
      VR_t[offset] = MU_t[data.num_nodes * patt.srcs[j] + i];
      VF_t[offset] = MU_t[data.num_nodes * patt.dsts[j] + i];
  };
  thrust::for_each_n(
    thrust::device,
    thrust::make_counting_iterator<Int>(0),
    data.num_nodes * patt.num_edges,
    init_VX
  );

  ac::RowMax2(patt.num_edges, data.num_nodes, VF_t, VFmax);
  ac::RowMax2(patt.num_edges, data.num_nodes, VR_t, VRmax);

  ac::EdgeMaxReduce2_t(
    data.num_edges, data.num_nodes, patt.num_edges,
    VFmax, RE_t, RMax_t, data.srcs
  );
  
  ac::EdgeMaxReduce2_t(
    data.num_edges, data.num_nodes, patt.num_edges,
    VRmax, FE_t, FMax_t, data.dsts
  );
  
  nvtxRangePop();
  auto prep_elapsed = prep_timer.stop();
  
  cuda_timer_t scatter_timer;
  scatter_timer.start();

  nvtxRangePushA("scatter");

  shard_n(CE_t, all_CE_t, n_gpus, patt.num_edges, data.num_edges, starts, ends);

  shard_n(FMax_t, all_FMax_t, n_gpus, patt.num_edges, data.num_nodes, starts, ends);
  shard_n(RMax_t, all_RMax_t, n_gpus, patt.num_edges, data.num_nodes, starts, ends);
  
  copy_n(data.srcs, all_data_srcs, n_gpus, data.num_edges, 1); // could use nccl
  copy_n(data.dsts, all_data_dsts, n_gpus, data.num_edges, 1);

  copy_n(patt.srcs, all_patt_srcs, n_gpus, patt.num_edges, 1);
  copy_n(patt.dsts, all_patt_dsts, n_gpus, patt.num_edges, 1);

  for(Int i = 0; i < n_gpus; i++) {
    hipSetDevice(i);
    hipDeviceSynchronize();
    hipSetDevice(0);
  }

  nvtxRangePop();
  auto scatter_elapsed = scatter_timer.stop();
  
  cuda_timer_t loop_timer;
  loop_timer.start();
  
  // --
  // Run

  for (Int i = 0; i < patt.num_nodes; i++) {
    nvtxRangePushA("loop");
    
    nvtxRangePushA("update_VX");
        
    // could fuse this into updateXMax_t
    #pragma omp parallel for num_threads(n_gpus)
    for(Int gid = 0; gid < n_gpus; gid++) {
      hipSetDevice(gid);
      
      Int start = starts[gid];
      Int end   = ends[gid];
      Int size  = end - start;
      
      Real* l_VF_t     = all_VF_t[gid];
      Real* l_VR_t     = all_VR_t[gid];
      Real* l_FMax_t   = all_FMax_t[gid];
      Real* l_RMax_t   = all_RMax_t[gid];
      Int* l_patt_srcs = all_patt_srcs[gid];
      Int* l_patt_dsts = all_patt_dsts[gid];
      
      auto update_VX = [=] __device__(Int const& offset) {
        Int r          = offset / data.num_nodes;
        Int c          = offset % data.num_nodes;
        l_VF_t[offset] = MU_t[data.num_nodes * l_patt_dsts[start + r] + c] - l_FMax_t[offset];
        l_VR_t[offset] = MU_t[data.num_nodes * l_patt_srcs[start + r] + c] - l_RMax_t[offset];
      };
      thrust::for_each_n(
        thrust::cuda::par.on(infos[gid].stream),
        thrust::make_counting_iterator<Int>(0),
        size * data.num_nodes,
        update_VX
      );
      hipEventRecord(infos[gid].event, infos[gid].stream);
    }

    nvtxRangePop();
    
    nvtxRangePushA("updateXMax_t");
    ac::updateXMax_t(
      patt.num_nodes, patt.num_edges, data.num_nodes, data.num_edges,
      all_CE_t,
      all_VF_t,
      all_VFmax,
      all_RE_t,
      all_RE_tmp,
      all_RMax_t,
      all_data_srcs,
      all_data_srcs,
      n_gpus,
      starts,
      ends,
      RMax_t,
      infos,
      F_max_reducers,
      F_sum_reducers
    );
    
    // could remove synchronization here
    
    ac::updateXMax_t(
      patt.num_nodes, patt.num_edges, data.num_nodes, data.num_edges,
      all_CE_t,
      all_VR_t,
      all_VRmax,
      all_FE_t,
      all_FE_tmp,
      all_FMax_t,
      all_data_srcs,
      all_data_dsts,
      n_gpus,
      starts,
      ends,
      FMax_t,
      infos,
      R_max_reducers,
      R_sum_reducers
    );
    
    for(Int gid = 0; gid < n_gpus; gid++)
        hipStreamWaitEvent(master_stream, infos[gid].event, 0);
    hipStreamSynchronize(master_stream);

    nvtxRangePop();
    
    nvtxRangePushA("ComputeMU2_t");
    // random row-write -- BAD
    ac::ComputeMU2_t(
      data.num_nodes, patt.num_edges, data.num_nodes, patt.num_nodes, // typo?
      CV_t,
      FMax_t,
      RMax_t,
      patt.srcs,
      patt.dsts,
      MU_t,
      master_stream
    );

    // simple row-wise -- OK
    ac::RowSoftmax2_prealloc(patt.num_nodes, data.num_nodes, MU_t, MU_tmp, master_stream);
    hipEventRecord(master_event, master_stream);
    hipStreamWaitEvent(master_stream, master_event, 0);

    nvtxRangePop();
    
    nvtxRangePop();
  }

  nvtxRangePop();
  auto loop_elapsed = loop_timer.stop();
  long long elapsed = timer.stop();
#if 1 
  std::cout 
    << "elapsed="            << elapsed 
    << " | prep_elapsed="    << prep_elapsed 
    << " | scatter_elapsed=" << scatter_elapsed 
    << " | loop_elapsed="    << loop_elapsed 
    << " | n_gpus="          << n_gpus
  << std::endl;
#endif

  // --
  // Copy results to host and print

#if 0
  ac::transpose(MU_t, MU, patt.num_nodes, data.num_nodes);
  Real *h_MU = (Real *) malloc(data.num_nodes * patt.num_nodes * sizeof(Real));
  hipMemcpy(h_MU, MU, data.num_nodes * patt.num_nodes * sizeof(Real), hipMemcpyDeviceToHost);
  for (Int i = 0; i < data.num_nodes * patt.num_nodes; i ++) printf("%e\n", h_MU[i]);
#endif



  auto j = gpu_info_json();

  // save the command line
  std::ostringstream command_line;
  for(int i = 0; i < argc; i++) {
      command_line << argv[i] << " ";
  }
  j["command-line"] = command_line.str();

  j["primitive"] = "ac";
  j["graph-file"] = {std::string(argv[1]), std::string(argv[2])};
  j["graph-edges"] = {data.num_edges, patt.num_edges};
  j["graph-nodes"] = {data.num_nodes, patt.num_edges};
  j["avg-process-time"] = (float)elapsed/1000;
  time_t now = time(NULL);
  j["time"] = ctime(&now);

  // get datat.bin name and pattern.bin name for variant
  auto data_file = std::string(argv[1]);
  auto s0 = data_file.find_last_of("/") + 1;
  auto s1 = data_file.find_last_of("_");
  data_file = data_file.substr(s0, s1-s0);
  auto pattern_file = std::string(argv[2]);
  s0 = pattern_file.find_last_of("/") + 1;
  s1 = pattern_file.find_last_of("_");
  pattern_file = pattern_file.substr(s0, s1-s0);
  j["tag"] = {std::string("variant:" + data_file + "-" + pattern_file), 
	      std::string("num-gpus:") + std::to_string(n_gpus)};

  // get the dataset from the json
  auto dataset = std::string(argv[3]);
  std::size_t p1 = dataset.find("ac__") + 4; // skip the expected "ac__"
  std::size_t p2 = dataset.find("__GPU");
  j["dataset"] = dataset.substr(p1, p2-p1);

  std::cout << '\n' << std::setw(4) << j << '\n';
  std::ofstream output_json(argv[3]);
  output_json << std::setw(4) << j << std::endl;

  return 0;
}
